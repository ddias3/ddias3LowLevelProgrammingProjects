#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <sys/time.h>

const int FACTORS = 30;

int numberMovies = 0;
int numberUsers = 0;

#define LEARNING_RATE (0.00125)
#define REGULARIZATION_PARAMETER (0.005)

#define BLOCKSIZE (32)
#define LEARNING_DATA_SIZE (99072112)
#define ERROR_CHANGE_THRESHOLD (100)
#define ERROR_THRESHOLD (0.01)//(0.145)
#define WARP_SIZE (32)

#define gpuErrorCheck(x) { gpuAssert((x), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char* fileName, int lineNumber, int abort = 1)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), fileName, lineNumber);
		if (abort)
			exit(code);
	}
}

typedef struct rating
{
	int rating;
	int userIndex;
	int movieIndex;
} rating_t;

rating_t* h_learningData;

double* h_userFactors;
double* h_movieFactors;

double get_walltime()
{
	struct timeval tp;
	gettimeofday(&tp, NULL);
	return (double)(tp.tv_sec + tp.tv_usec*1e-6);
}

void CreateLearningDataAndFactors()
{
	h_learningData = (rating_t*)malloc(sizeof(rating_t) * LEARNING_DATA_SIZE);
	assert(NULL != h_learningData);
	
	int indexLearningData = 0;
	FILE* netflixDataFile = fopen("netflix.dat", "r");
	char buffer[60];
	while (indexLearningData < LEARNING_DATA_SIZE && fgets(buffer, 60, netflixDataFile) != NULL)
	{
		assert(indexLearningData < LEARNING_DATA_SIZE);

		char* startPointer = buffer;
		char* endPointer = buffer;

		h_learningData[indexLearningData].userIndex = ((int)strtol(startPointer, &endPointer, 10)) - 1;
		startPointer = endPointer;
		h_learningData[indexLearningData].movieIndex = ((int)strtol(startPointer, &endPointer, 10)) - 1;
		startPointer = endPointer;
		h_learningData[indexLearningData].rating = ((int)strtol(startPointer, &endPointer, 10));

		if (h_learningData[indexLearningData].userIndex > numberUsers)
			numberUsers = h_learningData[indexLearningData].userIndex;
		if (h_learningData[indexLearningData].movieIndex > numberMovies)
			numberMovies = h_learningData[indexLearningData].movieIndex;
		
		++indexLearningData;
	}
	++numberMovies;
	++numberUsers;

	printf("Allocating memory for user and movie factors\n");

	h_userFactors = (double*)malloc(sizeof(double) * FACTORS * numberUsers);
	assert(NULL != h_userFactors);
	h_movieFactors = (double*)malloc(sizeof(double) * FACTORS * numberMovies);
	assert(NULL != h_movieFactors);

	#pragma omp parallel for
	for (int n = 0; n < (FACTORS * numberUsers); ++n)
		h_userFactors[n] = 1.0;

	int index = 0;
	int currentMovieIndex = 0;
	size_t sumRating = 0;
	size_t timesAdded = 0;
	while (index < LEARNING_DATA_SIZE)
	{
		if (h_learningData[index].movieIndex != currentMovieIndex)
		{
			for (int f = 0; f < FACTORS; ++f)
				h_movieFactors[currentMovieIndex * FACTORS + f] = (sumRating / ((double)timesAdded)) / ((double)FACTORS);

			currentMovieIndex = h_learningData[index].movieIndex;
			sumRating = 0;
			timesAdded = 0;
			continue;
		}

		sumRating += h_learningData[index].rating;
		++timesAdded;
		++index;
	}
	for (int f = 0; f < FACTORS; ++f)
		h_movieFactors[currentMovieIndex * FACTORS + f] = (sumRating / ((double)timesAdded)) / ((double)FACTORS);
	
	printf("Randomizing the learning data\n");

	srand(time(NULL));

	for (int n = 0; n < (LEARNING_DATA_SIZE / numberMovies); ++n)
	{
		rating_t tempRatingBlock;
		size_t m = ((rand() % numberMovies) * (LEARNING_DATA_SIZE / numberMovies)) + (rand() % (LEARNING_DATA_SIZE / numberMovies));
		assert(m < LEARNING_DATA_SIZE);

		tempRatingBlock.rating = h_learningData[n].rating;
		tempRatingBlock.userIndex = h_learningData[n].userIndex;
		tempRatingBlock.movieIndex = h_learningData[n].movieIndex;

		h_learningData[n].rating = h_learningData[m].rating;
		h_learningData[n].userIndex = h_learningData[m].userIndex;
		h_learningData[n].movieIndex = h_learningData[m].movieIndex;

		h_learningData[m].rating = tempRatingBlock.rating;
		h_learningData[m].userIndex = tempRatingBlock.userIndex;
		h_learningData[m].movieIndex = tempRatingBlock.movieIndex;
	}
}

__host__ __device__ double DotProduct(const double* vectorA, const double* vectorB, const int length)
{
	double dotProduct = 0.0;

	for (int n = 0; n < length; ++n)
		dotProduct += vectorA[n] * vectorB[n];

	return dotProduct;
}

__global__ void SingleIteration(const int learningDataLength, double* sumErrorSquared, rating_t* learningData, double* userFactors, double* movieFactors)
{
	//int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	//int index_y = blockIdx.y * blockDim.y + threadIdx.y;
	//int index = index_y * gridDim.x * blockDim.x + index_x;
	int indexKnownData = blockIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;
	if (indexKnownData == 0)
		*sumErrorSquared = 0.0;
	if (indexKnownData < learningDataLength)
	{
		int userIndex = learningData[indexKnownData].userIndex;
		int movieIndex = learningData[indexKnownData].movieIndex;
		int rating = learningData[indexKnownData].rating;

		double errorTerm = rating - DotProduct(&movieFactors[movieIndex * FACTORS], &userFactors[userIndex * FACTORS], FACTORS);
		//*sumErrorSquared += (errorTerm * errorTerm);

		for (int f = 0; f < FACTORS; ++f)
		{
			movieFactors[movieIndex * FACTORS + f] += LEARNING_RATE * (errorTerm * userFactors[userIndex * FACTORS + f]
				- REGULARIZATION_PARAMETER * movieFactors[movieIndex * FACTORS + f]);
		}

		for (int f = 0; f < FACTORS; ++f)
		{
			userFactors[userIndex * FACTORS + f] += LEARNING_RATE * (errorTerm * movieFactors[movieIndex * FACTORS + f]
				- REGULARIZATION_PARAMETER * userFactors[userIndex * FACTORS + f]);
		}
	}
}

__global__ void CalculateAggregateError(const int learningDataLength, double* sumErrorSquared, rating_t* learningData, double* userFactors, double* movieFactors)
{
	int indexKnownData = blockIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;
	if (indexKnownData < learningDataLength)
	{
		int userIndex = learningData[indexKnownData].userIndex;
		int movieIndex = learningData[indexKnownData].movieIndex;
		int rating = learningData[indexKnownData].rating;

		double errorTerm = rating - DotProduct(&movieFactors[movieIndex * FACTORS], &userFactors[userIndex * FACTORS], FACTORS);
		*sumErrorSquared += (errorTerm * errorTerm);
	}
}

void PrintPredictedData()
{
	int indexQueryData = 0;
	FILE* netflixQueryFile = fopen("netflix_query", "r");
	char buffer[60];
	while (fgets(buffer, 60, netflixQueryFile) != NULL)
	{
		char* startPointer = buffer;
		char* endPointer = buffer;

		int userIndex = ((int)strtol(startPointer, &endPointer, 10));
		startPointer = endPointer;
		int movieIndex = ((int)strtol(startPointer, &endPointer, 10));
		startPointer = endPointer;

		assert(userIndex < numberUsers);
		assert(movieIndex < numberMovies);

		double predictedRating = DotProduct(&h_movieFactors[movieIndex * FACTORS], &h_userFactors[userIndex * FACTORS], FACTORS);

		printf("User#:%d\tMovie#:%d\tPredicted Rating = %lf\n", userIndex, movieIndex, predictedRating);

		++indexQueryData;
	}
}

int main(int argc, char** argv)
{
	printf("Starting Netflix Recommender System Algorithm with Stochastic Gradient Descent\n");
	printf("Creating Learning Data\n");

	CreateLearningDataAndFactors();

	printf("Copying Data to GPU\n");

	double* h_sumErrorSquared = (double*)malloc(sizeof(double));
	double* d_sumErrorSquared;
	*h_sumErrorSquared = 0.0;
	gpuErrorCheck(hipMalloc((void**)&d_sumErrorSquared, sizeof(double)));
	gpuErrorCheck(hipMemcpy(d_sumErrorSquared, h_sumErrorSquared, sizeof(double), hipMemcpyHostToDevice));

	double* d_userFactors;
	double* d_movieFactors;
	rating_t* d_learningData;

	gpuErrorCheck(hipMalloc((void**)&d_userFactors, sizeof(double) * FACTORS * numberUsers));
	gpuErrorCheck(hipMalloc((void**)&d_movieFactors, sizeof(double) * FACTORS * numberMovies));
	gpuErrorCheck(hipMalloc((void**)&d_learningData, sizeof(rating_t) * LEARNING_DATA_SIZE));

	gpuErrorCheck(hipMemcpy(d_userFactors, h_userFactors, sizeof(double) * FACTORS * numberUsers, hipMemcpyHostToDevice));
	gpuErrorCheck(hipMemcpy(d_movieFactors, h_movieFactors, sizeof(double) * FACTORS * numberMovies, hipMemcpyHostToDevice));
	gpuErrorCheck(hipMemcpy(d_learningData, h_learningData, sizeof(rating_t) * LEARNING_DATA_SIZE, hipMemcpyHostToDevice));

	printf("Starting Algorith now\n");
	fflush(stdout);

	dim3 blockSize;
	blockSize.x = BLOCKSIZE;

	dim3 gridSize;

	int numberBlocksRequired = ((LEARNING_DATA_SIZE - 1) / BLOCKSIZE) + 1;

	if (numberBlocksRequired > 65535)
	{
		gridSize.x = 65535;
		gridSize.y = (numberBlocksRequired - 1) / 65535 + 1;
	}
	else
	{
		gridSize.x = numberBlocksRequired;
		gridSize.y = 1;
	}

	double previousErrorSquared = 9999999;
	double previousPreviousErrorSquared;

	unsigned int iterations = 0;
	double startTime = get_walltime();
	do
	{
		previousPreviousErrorSquared = previousErrorSquared;
		previousErrorSquared = *h_sumErrorSquared;

		SingleIteration<<<gridSize, blockSize>>>(LEARNING_DATA_SIZE, d_sumErrorSquared, d_learningData, d_userFactors, d_movieFactors);
		gpuErrorCheck(hipPeekAtLastError());
		gpuErrorCheck(hipDeviceSynchronize());

		CalculateAggregateError<<<gridSize, blockSize>>>(LEARNING_DATA_SIZE, d_sumErrorSquared, d_learningData, d_userFactors, d_movieFactors);
		gpuErrorCheck(hipPeekAtLastError());
		gpuErrorCheck(hipDeviceSynchronize());

		gpuErrorCheck(hipMemcpy(h_sumErrorSquared, d_sumErrorSquared, sizeof(double), hipMemcpyDeviceToHost));

		printf("\th_sumErrorSquared = %lf, (1/N(h_sumErrorSquared * WARP_SIZE))^(1/2) = %lf\n", *h_sumErrorSquared, sqrt(1.0 / LEARNING_DATA_SIZE * (*h_sumErrorSquared * WARP_SIZE)));
		++iterations;

		if (*h_sumErrorSquared > previousErrorSquared && previousErrorSquared > previousPreviousErrorSquared ||
			(abs(previousErrorSquared - *h_sumErrorSquared) + abs(previousPreviousErrorSquared - previousErrorSquared)) / 2.0 < ERROR_CHANGE_THRESHOLD)
			break;
	} while (*h_sumErrorSquared > (double)LEARNING_DATA_SIZE / (double)WARP_SIZE * (ERROR_THRESHOLD * ERROR_THRESHOLD));
	double endTime = get_walltime();
	
	printf("Performed %u iterations\n", iterations);

	printf("Total time = %lf\n", endTime - startTime);
	printf("Copying user and movie factors from GPU to host\n");

	gpuErrorCheck(hipMemcpy(h_userFactors, d_userFactors, sizeof(double) * FACTORS * numberUsers, hipMemcpyDeviceToHost));
	gpuErrorCheck(hipMemcpy(h_movieFactors, d_movieFactors, sizeof(double) * FACTORS * numberMovies, hipMemcpyDeviceToHost));

	PrintPredictedData();

	gpuErrorCheck(hipFree(d_sumErrorSquared));
	gpuErrorCheck(hipFree(d_userFactors));
	gpuErrorCheck(hipFree(d_movieFactors));
	gpuErrorCheck(hipFree(d_learningData));

	free(h_sumErrorSquared);
	free(h_userFactors);
	free(h_movieFactors);
	free(h_learningData);

	return 0;
}
